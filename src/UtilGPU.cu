#include "hip/hip_runtime.h"
#include "UtilGPU.cuh"
//#include ""
#include <stdio.h>
#include <base/time/Global_Timer.h>


template <class T>
__global__ void warpPerspectiveKernel(int in_rows,int in_cols,T* in_data,
                                      int out_rows,int out_cols,T* out_data,
                                      float* inv,T defVar)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;
        if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
        {
            out_data[x+y*out_cols]=in_data[(int)srcX+((int)srcY)*in_cols];
        }
        else
        {
            out_data[x+y*out_cols]=defVar;
        }
    }
}

template <class T>
bool operate<T>::warpPerspectiveCaller(int in_rows,int in_cols,T* in_data,
                           int out_rows,int out_cols,T* out_data,
                           float* inv,T defVar)
{
    T* in_dataGPU;
    T* out_dataGPU;
    float* invGPU;
    hipMalloc((void**) &in_dataGPU, in_cols*in_rows*sizeof(T));
    hipMalloc((void**) &out_dataGPU,out_cols*out_rows*sizeof(T));
    hipMalloc((void**) &invGPU,9*sizeof(float));
    hipMemcpy(in_dataGPU,in_data,in_cols*in_rows*sizeof(T),hipMemcpyHostToDevice);
    hipMemcpy(invGPU,inv,9*sizeof(float),hipMemcpyHostToDevice);

    dim3 threads(32,32);
        dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
//        dim3 grid(20,20);

    pi::timer.enter("warpPerspectiveKernel");
    warpPerspectiveKernel<T><<<grid,threads>>>(in_rows,in_cols,in_dataGPU,
                                               out_rows,out_cols,out_dataGPU,
                                               invGPU,defVar);
    pi::timer.leave("warpPerspectiveKernel");

    hipMemcpy(out_data,out_dataGPU,out_cols*out_rows*sizeof(T),hipMemcpyDeviceToHost);
    hipFree(in_dataGPU);hipFree(out_dataGPU);hipFree(invGPU);
    return true;
}

bool warpPerspective_uchar1(int in_rows,int in_cols,uchar1* in_data,
                            int out_rows,int out_cols,uchar1* out_data,
                            float* inv,uchar1 defVar)
{
    return operate<uchar1>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}
//bool warpPerspective_uchar2();
bool warpPerspective_uchar3(int in_rows,int in_cols,uchar3* in_data,
                            int out_rows,int out_cols,uchar3* out_data,
                            float* inv,uchar3 defVar)
{
    return operate<uchar3>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}

bool warpPerspective_uchar4(int in_rows,int in_cols,uchar4* in_data,
                            int out_rows,int out_cols,uchar4* out_data,
                            float* inv,uchar4 defVar)
{
    return operate<uchar4>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}


__global__ void renderFrameKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                  int out_rows,int out_cols,uchar4* out_data,
                                  bool fresh,uchar4 defVar,//image out
                                  float* inv,int centerX,int centerY//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        int idxOut=x+y*out_cols;

        // find source location
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;

        if(fresh) //warp
        {
            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                uchar4* ptrOut=out_data+idxOut;
                *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                    if(srcW<1) srcW=1;
                    //center weight
                }
                ptrOut->w=srcW;
            }
            else
            {
                out_data[idxOut]=defVar;
            }

        }
        else if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)// blender
        {
            uchar4* ptrOut=out_data+idxOut;
            // compute weight
            {
                //image weight
                float difX=srcX-in_rows*0.5;
                float difY=srcY-in_cols*0.5;
                srcW=1000*(0.25-(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols));
                if(srcW<1) srcW=1;
                //center weight
            }
            if(ptrOut->w<srcW)
            {
                ptrOut->w=srcW;
                uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                *((uchar3*)ptrOut)=*ptrIn;
            }
        }
    }
}

__global__ void renderFramesKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                   int out_rows,int out_cols,uchar4** out_datas,
                                   bool* freshs,uchar4 defVar,//image out
                                   float* invs,int* centers,int eleNum//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        for(int i=0;i<eleNum;i++)
        {
            uchar4* out_data=out_datas[i];
            bool    fresh=freshs[i];
            float*  inv=invs+9*i;

            int idxOut=x+y*out_cols;
            uchar4* ptrOut=out_data+idxOut;

            // find source location
            float srcX=inv[0]*x+inv[1]*y+inv[2];
            float srcY=inv[3]*x+inv[4]*y+inv[5];
            float srcW=inv[6]*x+inv[7]*y+inv[8];
            srcW=1./srcW;srcX*=srcW;srcY*=srcW;

            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=(0.25-(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols));//0~0.25
                    //center weight
                    if(1)
                    {
                        difX=centers[i*2]-x;
                        difY=centers[i*2+1]-y;
                        srcW=5e4*srcW/sqrt(difX*difX+difY*difY+1);
                    }
                    else
                        srcW=1000*srcW;
                    if(srcW<1) srcW=1;
                    else if(srcW>255) srcW=255;
                }
                if(fresh)
                {
                    *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];

                    ptrOut->w=srcW;
                }
                else// blender
                {
                    if(ptrOut->w<srcW)
                    {
                        ptrOut->w=srcW;
                        uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                        *((uchar3*)ptrOut)=*ptrIn;
                    }
                }
            }
            else if(fresh)
            {
                *ptrOut=defVar;
            }
        }
    }
}

__global__ void renderFramesKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                   int out_rows,int out_cols,float4** out_datas,
                                   bool* freshs,float4 defVar,//image out
                                   float* invs,int* centers,int eleNum//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        for(int i=0;i<eleNum;i++)
        {
            float4* out_data=out_datas[i];
            bool    fresh=freshs[i];
            float*  inv=invs+9*i;

            int idxOut=x+y*out_cols;
            float4* ptrOut=out_data+idxOut;

            // find source location
            float srcX=inv[0]*x+inv[1]*y+inv[2];
            float srcY=inv[3]*x+inv[4]*y+inv[5];
            float srcW=inv[6]*x+inv[7]*y+inv[8];
            srcW=1./srcW;srcX*=srcW;srcY*=srcW;

            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=(0.25-(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols));//0~0.25
                    //center weight
                    if(0)
                    {
                        difX=centers[i*2]-x;
                        difY=centers[i*2+1]-y;
                        srcW=1e5*srcW/(difX*difX+difY*difY+1000);
                    }
                }
                if(fresh||ptrOut->w<srcW)
                {
                    uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                    ptrOut->x=ptrIn->x*0.00392f;
                    ptrOut->y=ptrIn->y*0.00392f;
                    ptrOut->z=ptrIn->z*0.00392f;
                    ptrOut->w=srcW;
                }
            }
            else if(fresh)
            {
                *ptrOut=defVar;
            }
        }
    }
}

bool renderFrameCaller(CudaImage<uchar3>& rgbIn,CudaImage<uchar4>& ele,
                       float* inv,int centerX,int centerY)
{
    float* invGPU;
    checkCudaErrors(hipMalloc((void**) &invGPU,9*sizeof(float)));
    checkCudaErrors(hipMemcpy(invGPU,inv,9*sizeof(float),hipMemcpyHostToDevice));
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(ele.cols, threads.x), divUp(ele.rows, threads.y));
    renderFrameKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        ele.rows,ele.cols,ele.data,
                                        ele.fresh,defVar,invGPU,centerX,centerY);
    checkCudaErrors(hipFree(invGPU));
    return true;
}


bool renderFramesCaller(CudaImage<uchar3>& rgbIn,int out_rows,int out_cols,
                        uchar4** out_datas,bool* freshs,
                       float* invs,int* centers,int eleNum)
{
    float* invGPU;
    uchar4** outDataGPU;
    bool*  freshesGPU;
    int*   centersGPU;

    checkCudaErrors(hipMalloc((void**) &invGPU,9*sizeof(float)*eleNum));
    checkCudaErrors(hipMalloc((void**) &outDataGPU,sizeof(uchar4*)*eleNum));
    checkCudaErrors(hipMalloc((void**) &freshesGPU,sizeof(bool)*eleNum));
    checkCudaErrors(hipMalloc((void**) &centersGPU,2*sizeof(int)*eleNum));
    checkCudaErrors(hipMemcpy(invGPU,invs,9*sizeof(float)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(outDataGPU,out_datas,sizeof(uchar4*)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(freshesGPU,freshs,sizeof(bool)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(centersGPU,centers,2*sizeof(int)*eleNum,hipMemcpyHostToDevice));
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
    renderFramesKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        out_rows,out_cols,outDataGPU,freshesGPU,
                                        defVar,invGPU,centersGPU,eleNum);
    checkCudaErrors(hipFree(invGPU));
    checkCudaErrors(hipFree(outDataGPU));
    checkCudaErrors(hipFree(freshesGPU));
    checkCudaErrors(hipFree(centersGPU));
    return true;
}

bool renderFramesCaller(CudaImage<uchar3>& rgbIn,int out_rows,int out_cols,
                        float4** out_datas,bool* freshs,
                       float* invs,int* centers,int eleNum)
{
    float* invGPU;
    float4** outDataGPU;
    bool*  freshesGPU;
    int*   centersGPU;

    checkCudaErrors(hipMalloc((void**) &invGPU,9*sizeof(float)*eleNum));
    checkCudaErrors(hipMalloc((void**) &outDataGPU,sizeof(float4*)*eleNum));
    checkCudaErrors(hipMalloc((void**) &freshesGPU,sizeof(bool)*eleNum));
    checkCudaErrors(hipMalloc((void**) &centersGPU,2*sizeof(int)*eleNum));
    checkCudaErrors(hipMemcpy(invGPU,invs,9*sizeof(float)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(outDataGPU,out_datas,sizeof(float4*)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(freshesGPU,freshs,sizeof(bool)*eleNum,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(centersGPU,centers,2*sizeof(int)*eleNum,hipMemcpyHostToDevice));
    dim3 threads(32,32);
    float4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
    renderFramesKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        out_rows,out_cols,outDataGPU,freshesGPU,
                                        defVar,invGPU,centersGPU,eleNum);
    checkCudaErrors(hipFree(invGPU));
    checkCudaErrors(hipFree(outDataGPU));
    checkCudaErrors(hipFree(freshesGPU));
    checkCudaErrors(hipFree(centersGPU));
    return true;
}


