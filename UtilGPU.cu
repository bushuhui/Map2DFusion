#include "hip/hip_runtime.h"
#include "UtilGPU.cuh"
#include ""
#include <stdio.h>
#include <base/time/Global_Timer.h>


template <class T>
void __global__ addKernel1(T *c, const T *a, const T *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

template <class T>
hipError_t operate<T>::addWithCuda(T *c, const T *a, const T *b, unsigned int size)
{
    T *dev_a = 0;
    T *dev_b = 0;
    T *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(T));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(T));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(T));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(T), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(T), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel1<T><<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(T), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}

extern "C" void runtest()
{
    const int arraySize = 5;
    const double a_d[arraySize] = { 1.1, 2.2, 3.3, 4.4, 5.5 };
    const double b_d[arraySize] = { 10.1, 20.1, 30.1, 40.1, 50.1 };
    double c_d[arraySize] = { 0 };

    // Add vectors in parallel.
    operate<double> op;
    hipError_t cudaStatus = op.addWithCuda(c_d, a_d, b_d, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return;
    }

    printf("{1.1,2.2,3.3,4.4,5.5} + {10.1,20.1,30.1,40.1,50.1} = {%f,%f,%f,%f,%f}\n",
        c_d[0], c_d[1], c_d[2], c_d[3], c_d[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return;
    }
}

template <class T>
__global__ void warpPerspectiveKernel(int in_rows,int in_cols,T* in_data,
                                      int out_rows,int out_cols,T* out_data,
                                      float* inv,T defVar)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < out_rows &&  x< out_cols)
    {
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;
        if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
        {
            out_data[x+y*out_cols]=in_data[(int)srcX+((int)srcY)*in_cols];
        }
        else
        {
            out_data[x+y*out_cols]=defVar;
        }
    }
}

template <class T>
bool operate<T>::warpPerspectiveCaller(int in_rows,int in_cols,T* in_data,
                           int out_rows,int out_cols,T* out_data,
                           float* inv,T defVar)
{
    T* in_dataGPU;
    T* out_dataGPU;
    float* invGPU;
    hipMalloc((void**) &in_dataGPU, in_cols*in_rows*sizeof(T));
    hipMalloc((void**) &out_dataGPU,out_cols*out_rows*sizeof(T));
    hipMalloc((void**) &invGPU,9*sizeof(float));
    hipMemcpy(in_dataGPU,in_data,in_cols*in_rows*sizeof(T),hipMemcpyHostToDevice);
    hipMemcpy(invGPU,inv,9*sizeof(float),hipMemcpyHostToDevice);

    dim3 threads(32,32);
        dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
//        dim3 grid(20,20);

    pi::timer.enter("warpPerspectiveKernel");
    warpPerspectiveKernel<T><<<grid,threads>>>(in_rows,in_cols,in_dataGPU,
                                               out_rows,out_cols,out_dataGPU,
                                               invGPU,defVar);
    pi::timer.leave("warpPerspectiveKernel");

    hipMemcpy(out_data,out_dataGPU,out_cols*out_rows*sizeof(T),hipMemcpyDeviceToHost);
    hipFree(in_dataGPU);hipFree(out_dataGPU);hipFree(invGPU);
    return true;
}

bool warpPerspective_uchar1(int in_rows,int in_cols,uchar1* in_data,
                            int out_rows,int out_cols,uchar1* out_data,
                            float* inv,uchar1 defVar)
{
    return operate<uchar1>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}
//bool warpPerspective_uchar2();
bool warpPerspective_uchar3(int in_rows,int in_cols,uchar3* in_data,
                            int out_rows,int out_cols,uchar3* out_data,
                            float* inv,uchar3 defVar)
{
    return operate<uchar3>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}

bool warpPerspective_uchar4(int in_rows,int in_cols,uchar4* in_data,
                            int out_rows,int out_cols,uchar4* out_data,
                            float* inv,uchar4 defVar)
{
    return operate<uchar4>::warpPerspectiveCaller(in_rows,in_cols,in_data,
                                                  out_rows,out_cols,out_data,
                                                  inv,defVar);
}


__global__ void renderFrameKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                  int out_rows,int out_cols,uchar4* out_data,
                                  bool fresh,uchar4 defVar,//image out
                                  float* inv,int centerX,int centerY//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        int idxOut=x+y*out_cols;

        // find source location
        float srcX=inv[0]*x+inv[1]*y+inv[2];
        float srcY=inv[3]*x+inv[4]*y+inv[5];
        float srcW=inv[6]*x+inv[7]*y+inv[8];
        srcW=1./srcW;srcX*=srcW;srcY*=srcW;

        if(fresh) //warp
        {
            if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
            {
                uchar4* ptrOut=out_data+idxOut;
                *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                    if(srcW<1) srcW=1;
                    //center weight
                }
                ptrOut->w=srcW;
            }
            else
            {
                out_data[idxOut]=defVar;
            }

        }
        else if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)// blender
        {
            uchar4* ptrOut=out_data+idxOut;
            // compute weight
            {
                //image weight
                float difX=srcX-in_rows*0.5;
                float difY=srcY-in_cols*0.5;
                srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                if(srcW<1) srcW=1;
                srcW=1;
                //center weight
            }
            if(ptrOut->w<srcW)
            {
                ptrOut->w=srcW;
                uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                *((uchar3*)ptrOut)=*ptrIn;
            }
        }
    }
}

__global__ void renderFramesKernel(int in_rows,int in_cols,uchar3* in_data,//image in
                                   int out_rows,int out_cols,uchar4** out_datas,
                                   bool* freshs,uchar4 defVar,//image out
                                   float* invs,int centerX,int centerY,int eleNum//relations
                                  )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(y<out_rows&&x<out_cols)
    {
        for(int i=0;i<eleNum;i++)
        {
            uchar4* out_data=out_datas[i];
            bool    fresh=freshs[i];
            float*  inv=invs+9*i;

            int idxOut=x+y*out_cols;

            // find source location
            float srcX=inv[0]*x+inv[1]*y+inv[2];
            float srcY=inv[3]*x+inv[4]*y+inv[5];
            float srcW=inv[6]*x+inv[7]*y+inv[8];
            srcW=1./srcW;srcX*=srcW;srcY*=srcW;

            if(fresh) //warp
            {
                if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)
                {
                    uchar4* ptrOut=out_data+idxOut;
                    *((uchar3*)ptrOut)=in_data[(int)srcX+((int)srcY)*in_cols];
                    // compute weight
                    {
                        //image weight
                        float difX=srcX-in_rows*0.5;
                        float difY=srcY-in_cols*0.5;
                        srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                        if(srcW<1) srcW=1;
                        //center weight
                    }
                    ptrOut->w=srcW;
                }
                else
                {
                    out_data[idxOut]=defVar;
                }

            }
            else if(srcX<in_cols&&srcX>=0&&srcY<in_rows&&srcY>=0)// blender
            {
                uchar4* ptrOut=out_data+idxOut;
                // compute weight
                {
                    //image weight
                    float difX=srcX-in_rows*0.5;
                    float difY=srcY-in_cols*0.5;
                    srcW=1000*(difX*difX+difY*difY)/(in_rows*in_rows+in_cols*in_cols);
                    if(srcW<1) srcW=1;
                    srcW=1;
                    //center weight
                }
                if(ptrOut->w<srcW)
                {
                    ptrOut->w=srcW;
                    uchar3* ptrIn =in_data +(int)srcX+((int)srcY)*in_cols;
                    *((uchar3*)ptrOut)=*ptrIn;
                }
            }
        }
    }
}

bool renderFrameCaller(CudaImage<uchar3>& rgbIn,CudaImage<uchar4>& ele,
                       float* inv,int centerX,int centerY)
{
    float* invGPU;
    hipMalloc((void**) &invGPU,9*sizeof(float));
    hipMemcpy(invGPU,inv,9*sizeof(float),hipMemcpyHostToDevice);
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(ele.cols, threads.x), divUp(ele.rows, threads.y));
    renderFrameKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        ele.rows,ele.cols,ele.data,
                                        ele.fresh,defVar,invGPU,centerX,centerY);
    hipFree(invGPU);
    return true;
}


bool renderFramesCaller(CudaImage<uchar3>& rgbIn,int out_rows,int out_cols,
                        uchar4** out_datas,bool* freshs,
                       float* invs,int centerX,int centerY,int eleNum)
{
    float* invGPU;
    uchar4** outDataGPU;
    bool*  freshesGPU;

    hipMalloc((void**) &invGPU,9*sizeof(float)*eleNum);
    hipMalloc((void**) &outDataGPU,sizeof(uchar4*)*eleNum);
    hipMalloc((void**) &freshesGPU,sizeof(bool)*eleNum);
    hipMemcpy(invGPU,invs,9*sizeof(float)*eleNum,hipMemcpyHostToDevice);
    hipMemcpy(outDataGPU,out_datas,sizeof(uchar4*)*eleNum,hipMemcpyHostToDevice);
    hipMemcpy(freshesGPU,freshs,sizeof(bool)*eleNum,hipMemcpyHostToDevice);
    dim3 threads(32,32);
    uchar4 defVar;
    defVar.x=defVar.y=defVar.z=defVar.w=0;
    dim3 grid(divUp(out_cols, threads.x), divUp(out_rows, threads.y));
    renderFramesKernel<<<grid,threads>>>(rgbIn.rows,rgbIn.cols,rgbIn.data,
                                        out_rows,out_cols,outDataGPU,freshesGPU,
                                        defVar,invGPU,centerX,centerY,eleNum);
    hipFree(invGPU);
    hipFree(outDataGPU);
    hipFree(freshesGPU);
    return true;
}



